
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdlib.h>
#include<stdio.h>
#include <chrono>       /* measure time */


__global__ void gpuAdd(float* a, float* b, float* c, int N){

        int threadId = threadIdx.x + blockIdx.x * blockDim.x;
        int stride = blockDim.x * gridDim.x;

        for (int idx=threadId; idx<N; idx += stride){
                c[idx] = a[idx] + b[idx];
        }

        //std::cout << "blockDim.x: " << blockDim.x << std::endl;
        //std::cout << "gridDim.x: " << gridDim.x << std::endl;
}



int main(){
        // declare host memory and put data in
        int N = 50550;
        float* h_a, *h_b, *h_c;

        // allocate memory
        h_a = (float *)malloc(N*sizeof(float));
        h_b = (float *)malloc(N*sizeof(float));
        h_c = (float *)malloc(N*sizeof(float));

        // init data
        for (int i=0; i<N; ++i){
                h_a[i] = (float)N-i-1;
                h_b[i] = (float)i-N+i+2;
        }

        // declare device memory
        float* d_a, *d_b, *d_c;

        hipMalloc((float**)&d_a, N*sizeof(float));
        hipMalloc((float**)&d_b, N*sizeof(float));
        hipMalloc((float**)&d_c, N*sizeof(float));

        // start the clock
        std::chrono::high_resolution_clock::time_point t11 = std::chrono::high_resolution_clock::now();

        // copy data to device
        hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_a, h_b, N*sizeof(float), hipMemcpyHostToDevice);

        // use device function
        gpuAdd<<<16, 64>>>(d_a, d_b, d_c, N);

        hipDeviceSynchronize();

        // copy result from device back to host
        hipMemcpy(h_c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);


        // measure time 
        std::chrono::high_resolution_clock::time_point t12 = std::chrono::high_resolution_clock::now();

        // reduce to sum
        float s = 0.0;
        for (int i=0; i<N; ++i){
                s+=h_c[i];
        }

        // free up device memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        // free host memory
        free(h_a);
        free(h_b);
        free(h_c);

        //output
        std::cout << "sum: "<< s << std::endl;

        // Time measurements
        std::cout << "cudaMemcpy: host->device: " << std::chrono::duration_cast<std::chrono::milliseconds>(t12 - t11).count() << " ms\n" << std::endl;
}
