

int main(){
        // declare host memory and put data in
        int N = 1024;
        float* h_a, *h_b, *h_c;

        // allocate memory
        h_a = (float *)malloc(N*sizeof(float));
        h_b = (float *)malloc(N*sizeof(float));
        h_c = (float *)malloc(N*sizeof(float));

        // init data
        for (int i=0; i<N; ++i){
                h_a[i] = (float)N-i-1;
                h_b[i] = (float)i-N+i+2;
        }

        // declare device memory
        float* d_a, *d_b, *d_c;

        hipMalloc((float**)&d_a, N*sizeof(float));
        hipMalloc((float**)&d_b, N*sizeof(float));
        hipMalloc((float**)&d_c, N*sizeof(float));

        // copy data to device
        hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_a, h_b, N*sizeof(float), hipMemcpyHostToDevice);

        // use device function
        gpuAdd<<<16, 64>>>(d_a, d_b, d_c, N);

        hipDeviceSynchronize();

        // copy result from device back to host
        hipMemcpy(h_c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);

        // reduce to sum
        float s = 0.0;
        for (int i=0; i<N; ++i){
                s+=h_c[i];
        }

        // free up device memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        // free host memory
        free(h_a);
        free(h_b);
        free(h_c);


        std::cout << "Sum: " << s << std::endl;


        // free up memory
        free(h_a);
        free(h_b);
        free(h_c);
}